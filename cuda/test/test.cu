#include <limits>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "hip/hip_runtime.h"


#define TEST_INPUT_NUM 80

float input[TEST_INPUT_NUM] = {
    -55.795, -42.349, 79.255,  5.941,   96.018,  31.294,  -96.905, 53.291,
    -90.021, -11.393, 57.446,  41.810,  25.299,  98.622,  37.640,  -11.657,
    55.496,  45.014,  97.440,  -65.244, 20.372,  -40.049, 57.645,  -16.184,
    -95.877, -94.809, 64.730,  -60.664, -44.394, 51.183,  48.071,  -84.458,
    30.944,  -42.142, 7.816,   -22.595, -89.318, -6.443,  42.550,  81.024,
    53.816,  80.649,  76.768,  71.688,  -25.017, -36.421, 37.996,  70.614,
    -73.219, 38.254,  -58.320, -4.735,  -54.836, -94.747, -36.710, 98.710,
    79.415,  -13.906, -61.225, -24.768, -96.095, -88.033, 12.208,  36.603,
    80.426,  -15.617, -28.778, 35.831,  42.338,  -19.250, 89.067,  -44.727,
    80.168,  -77.544, -60.959, -78.450, -74.932, 20.722,  -22.494, 55.845};

float output[TEST_INPUT_NUM];

__global__ void test_inc_kernel(float *input_gpu, float *output_gpu,
                                int32_t num_items) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_items) {
    output_gpu[idx] = input_gpu[idx] + 1.0f;
  }
}

__global__ void test_inc_kernel2(float *input_gpu, float *output_gpu,
                                 int32_t num_items) {
  int block_size =
      (num_items + (blockDim.x * gridDim.x) - 1) / (blockDim.x * gridDim.x);
  for (int32_t i = 0; i < block_size; i++) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * block_size + i;
    if (idx < num_items) {
      output_gpu[idx] = input_gpu[idx] + 1.0f;
    }
  }
}

void print_fvec(float* vec, int32_t num_items) {
  for (int32_t i = 0; i < num_items; i++) {
    printf("%f, ", (float)(vec[i]));
    if (i % 10 == 9) {
      printf("\n");
    }
  }
}

void test_inc() {
  float *input_gpu;
  hipMalloc(&input_gpu, sizeof(float) * TEST_INPUT_NUM);
  float *output_gpu;
  hipMalloc(&output_gpu, sizeof(float) * TEST_INPUT_NUM);
  hipMemcpy(input_gpu, input, sizeof(float) * TEST_INPUT_NUM,
             hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16);
  dim3 numBlocks((TEST_INPUT_NUM + threadsPerBlock.x - 1) / threadsPerBlock.x);
  printf("numBlocks: %d, threadsPerBlock: %d\n", numBlocks.x,
         threadsPerBlock.x);
  test_inc_kernel<<<numBlocks, threadsPerBlock>>>(input_gpu, output_gpu,
                                                  TEST_INPUT_NUM);

  hipMemcpy(output, output_gpu, sizeof(float) * TEST_INPUT_NUM,
             hipMemcpyDeviceToHost);

  print_fvec(output, TEST_INPUT_NUM);

  hipFree(input_gpu);
  hipFree(output_gpu);
}

void test_inc2() {
  float *input_gpu;
  hipMalloc(&input_gpu, sizeof(float) * TEST_INPUT_NUM);
  float *output_gpu;
  hipMalloc(&output_gpu, sizeof(float) * TEST_INPUT_NUM);
  hipMemcpy(input_gpu, input, sizeof(float) * TEST_INPUT_NUM,
             hipMemcpyHostToDevice);

  dim3 threadsPerBlock(2);
  dim3 numBlocks(4);
  printf("numBlocks: %d, threadsPerBlock: %d\n", numBlocks.x,
         threadsPerBlock.x);
  test_inc_kernel2<<<numBlocks, threadsPerBlock>>>(input_gpu, output_gpu,
                                                   79);

  hipMemcpy(output, output_gpu, sizeof(float) * TEST_INPUT_NUM,
             hipMemcpyDeviceToHost);

  print_fvec(output, TEST_INPUT_NUM);

  hipFree(input_gpu);
  hipFree(output_gpu);
}

int main(){

    // test_inc();
    test_inc2();
    hipDeviceSynchronize();
}