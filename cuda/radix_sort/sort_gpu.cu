#include "hip/hip_runtime.h"
#include <limits>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "sort_gpu.cuh"

float input[TEST_INPUT_NUM] = {
    -55.795, -42.349, 79.255,  5.941,   96.018,  31.294,  -96.905, 53.291,
    -90.021, -11.393, 57.446,  41.810,  25.299,  98.622,  37.640,  -11.657,
    55.496,  45.014,  97.440,  -65.244, 20.372,  -40.049, 57.645,  -16.184,
    -95.877, -94.809, 64.730,  -60.664, -44.394, 51.183,  48.071,  -84.458,
    30.944,  -42.142, 7.816,   -22.595, -89.318, -6.443,  42.550,  81.024,
    53.816,  80.649,  76.768,  71.688,  -25.017, -36.421, 37.996,  70.614,
    -73.219, 38.254,  -58.320, -4.735,  -54.836, -94.747, -36.710, 98.710,
    79.415,  -13.906, -61.225, -24.768, -96.095, -88.033, 12.208,  36.603,
    80.426,  -15.617, -28.778, 35.831,  42.338,  -19.250, 89.067,  -44.727,
    80.168,  -77.544, -60.959, -78.450, -74.932, 20.722,  -22.494, 55.845};

float output[TEST_INPUT_NUM];
int32_t indices_ptr_out[TEST_INPUT_NUM];

#if 0
int32_t keys[TEST_INPUT_NUM] = {0};
int32_t bfe_keys[TEST_INPUT_NUM] = {0};
int32_t bfe_keys_out[TEST_INPUT_NUM] = {0};
int32_t offset[TEST_INPUT_NUM] = {0};
// int32_t bucket_offset[BUCKET_SIZE] = {0};
// int32_t curr_count[THREAD_NUM] = {0};
int32_t bucket_offset[BUCKET_SIZE][THREAD_NUM] = {0};
int32_t exclusive_cumsum[BUCKET_SIZE][THREAD_NUM] = {0};
// int32_t exclusive_cumsum[BUCKET_SIZE] = {0};
int32_t indices[2][TEST_INPUT_NUM];
#endif

__global__ void prepare_indices(int32_t *indices, int32_t num_items) {
  int block_size =
      num_items + (blockDim.x * gridDim.x) - 1 / (blockDim.x * gridDim.x);
  for (int32_t i = 0; i < block_size; i++) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * block_size + i;
    if (idx < num_items) {
      indices[idx] = idx;
    }
  }
}

#if 0
template <typename KeyT>
void extract_keys(KeyT *d_keys_in, KeyT *d_keys_out, int32_t *indices,
                  int32_t num_items, int32_t bit_start, int32_t num_bits) {
  for (int32_t i = 0; i < num_items; i++) {
    d_keys_out[i] = Unsigned_Bits<KeyT>::BitfieldExtract(d_keys_in[indices[i]],
                                                         bit_start, num_bits);
  }
}

void calc_exclusive_cumsum(const int32_t *value_in, int32_t *exclusive_cumsum,
                           int32_t num_items) {
  int32_t sum;
  for (int32_t i = 0; i < num_items; i++) {
    if (i == 0) {
      sum = 0;
    } else {
      sum += value_in[i - 1];
    }
    exclusive_cumsum[i] = sum;
  }
}

void update_indices_ptr(const int32_t *d_keys_in, const int32_t *indices_ptr_in,
                        const int32_t *offset, const int32_t *exclusive_cumsum,
                        int32_t *indices_ptr_out, int32_t num_items) {
  int32_t num_items_per_thread = num_items / THREAD_NUM;
  for (int32_t i = 0; i < THREAD_NUM; i++) {
    for (int32_t j = 0; j < num_items_per_thread; j++) {
      int32_t idx0 = j + i * num_items_per_thread;
      int32_t idx = offset[idx0] + exclusive_cumsum[d_keys_in[idx0] * THREAD_NUM + i];
      indices_ptr_out[idx] = indices_ptr_in[idx0];
      bfe_keys_out[idx] = bfe_keys[idx0];
    }
  }
}

void sort_pairs_loop(const int32_t *d_keys_in, int32_t *indices_ptr_in,
                     int32_t *indices_ptr_out, int32_t num_items) {
  int32_t num_items_per_thread = num_items / THREAD_NUM;
  for (int32_t i = 0; i < THREAD_NUM; i++) {
    for (int32_t j = 0; j < num_items_per_thread; j++) {
      int32_t idx = j + i * num_items_per_thread;
      offset[idx] = bucket_offset[d_keys_in[idx]][i];
      bucket_offset[d_keys_in[idx]][i]++;
    }
  }
  calc_exclusive_cumsum((int32_t *)bucket_offset, (int32_t *)exclusive_cumsum,
                        BUCKET_SIZE * THREAD_NUM);
  update_indices_ptr(d_keys_in, indices_ptr_in, offset, (int32_t *)exclusive_cumsum,
                     indices_ptr_out, num_items);
}

template <typename ValueT>
void post_process(const ValueT *d_values_in, ValueT *d_values_out,
                  int32_t *indices_ptr, int32_t *indices_ptr_out,
                  int32_t num_items) {
  for (int32_t i = 0; i < num_items; i++) {
    indices_ptr_out[i] = indices_ptr[i];
    d_values_out[i] = d_values_in[indices_ptr[i]];
  }
}

template <typename KeyT, typename ValueT, bool is_descend>
void sort_pairs(const ValueT *d_values_in, ValueT *d_values_out,
                int32_t *indices_ptr, int32_t num_items) {
  int32_t loop_count = sizeof(KeyT) * 8 / BUCKET_WIDTH;
  KeyT *keys;
  hipMalloc(&keys, sizeof(KeyT) * num_items);
  KeyT *bfe_keys;
  hipMalloc(&bfe_keys, sizeof(KeyT) * num_items);
  int32_t *offset;
  hipMalloc(&offset, sizeof(int32_t) * num_items);
  int32_t *bucket_offset;
  hipMalloc(&offset, sizeof(int32_t) * BUCKET_SIZE * THREAD_NUM);
  int32_t *exclusive_cumsum;
  hipMalloc(&exclusive_cumsum, sizeof(int32_t) * BUCKET_SIZE * THREAD_NUM);
  int32_t* indices_ptr_backup;
  hipMalloc(&indices_ptr_backup, sizeof(int32_t) * TEST_INPUT_NUM);

  prepare_keys<KeyT, ValueT, is_descend>(d_values_in, keys, num_items);
  prepare_indices(indices[0], num_items);
  for (int32_t i = 0; i < loop_count; i++) {
    int32_t begin_bit = (i)*BUCKET_WIDTH;
    extract_keys(keys, bfe_keys, indices[i % 2], num_items, begin_bit,
                 BUCKET_WIDTH);
    sort_pairs_loop(bfe_keys, indices[i % 2], indices[(i + 1) % 2], num_items);
    memset(bucket_offset, 0, sizeof(int32_t) * BUCKET_SIZE * THREAD_NUM);
  }
  post_process(d_values_in, d_values_out, indices[0], indices_ptr, num_items);

  hipFree(keys);
  hipFree(bfe_keys);
  hipFree(offset);
  hipFree(bucket_offset);
  hipFree(exclusive_cumsum);
  hipFree(indices_ptr_backup);
}
#endif

void test_sort() {
  DType *input_gpu;
  hipMalloc(&input_gpu, sizeof(float) * TEST_INPUT_NUM);
  DType *output_gpu;
  hipMalloc(&output_gpu, sizeof(float) * TEST_INPUT_NUM);
  int32_t *indices_ptr_out_gpu;
  hipMalloc(&indices_ptr_out_gpu, sizeof(int32_t) * TEST_INPUT_NUM);

  hipMemcpy(input_gpu, input, sizeof(float) * TEST_INPUT_NUM,
             hipMemcpyHostToDevice);

  // dim3 threadsPerBlock(THREAD_NUM);
  // dim3 numBlocks(2);
  // sort_pairs<int32_t, float, false>(input, output, indices_ptr_out,
  //                                   TEST_INPUT_NUM);

  hipMemcpy(output, output_gpu, sizeof(float) * TEST_INPUT_NUM,
             hipMemcpyDeviceToHost);
  hipMemcpy(indices_ptr_out, indices_ptr_out_gpu,
             sizeof(int32_t) * TEST_INPUT_NUM, hipMemcpyDeviceToHost);

  hipFree(input_gpu);
  hipFree(output_gpu);
  hipFree(indices_ptr_out);
}

int main() {
  // test_key<<<1, 2>>>();
  // test_sort();
  // test_prepare_keys();
  test_prepare_indices();
  hipDeviceSynchronize();
  return 0;
}
