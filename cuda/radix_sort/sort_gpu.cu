#include "hip/hip_runtime.h"
#include "sort_gpu.cuh"
#include <limits>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

float input[TEST_INPUT_NUM] = {
    -55.795, -42.349, 79.255,  5.941,   96.018,  31.294,  -96.905, 53.291,
    -90.021, -11.393, 57.446,  41.810,  25.299,  98.622,  37.640,  -11.657,
    55.496,  45.014,  97.440,  -65.244, 20.372,  -40.049, 57.645,  -16.184,
    -95.877, -94.809, 64.730,  -60.664, -44.394, 51.183,  48.071,  -84.458,
    30.944,  -42.142, 7.816,   -22.595, -89.318, -6.443,  42.550,  81.024,
    53.816,  80.649,  76.768,  71.688,  -25.017, -36.421, 37.996,  70.614,
    -73.219, 38.254,  -58.320, -4.735,  -54.836, -94.747, -36.710, 98.710,
    79.415,  -13.906, -61.225, -24.768, -96.095, -88.033, 12.208,  36.603,
    80.426,  -15.617, -28.778, 35.831,  42.338,  -19.250, 89.067,  -44.727,
    80.168,  -77.544, -60.959, -78.450, -74.932, 20.722,  -22.494, 55.845};

int32_t input_i[TEST_INPUT_NUM] = {
    -27, -78, 74,  -3,  -63, -70, -89, 96,  -8,  -71, 33,  1,   -44, -75,
    67,  -28, -22, -84, -40, -34, -47, 83,  -49, -5,  60,  -11, -8,  -48,
    -52, 58,  49,  76,  -24, -47, -31, 47,  -84, -86, 97,  30,  60,  41,
    -46, 66,  53,  -37, -28, 99,  -21, 78,  -30, -57, -88, 4,   55,  85,
    41,  79,  97,  -43, 60,  27,  90,  88,  -57, 33,  -60, 90,  44,  7,
    -18, -75, -72, 7,   -25, 34,  59,  -35, -13, 58};

float output[TEST_INPUT_NUM];
int32_t indices_ptr_out[TEST_INPUT_NUM];

#if 0
int32_t keys[TEST_INPUT_NUM] = {0};
int32_t bfe_keys[TEST_INPUT_NUM] = {0};
int32_t bfe_keys_out[TEST_INPUT_NUM] = {0};
int32_t offset[TEST_INPUT_NUM] = {0};
// int32_t bucket_offset[BUCKET_SIZE] = {0};
// int32_t curr_count[THREAD_NUM] = {0};
int32_t bucket_offset[BUCKET_SIZE][THREAD_NUM] = {0};
int32_t exclusive_cumsum[BUCKET_SIZE][THREAD_NUM] = {0};
// int32_t exclusive_cumsum[BUCKET_SIZE] = {0};
int32_t indices[2][TEST_INPUT_NUM];
#endif

__global__ void put_numbers_into_bucket(const int32_t *d_keys_in,
                                        int32_t *offset, int32_t *bucket_offset,
                                        int32_t num_items) {
  int32_t block_size =
      (num_items + (blockDim.x * gridDim.x) - 1) / (blockDim.x * gridDim.x);

  for (int32_t i = 0; i < block_size; i++) {
    int32_t idx = i + (blockIdx.x * blockDim.x + threadIdx.x) * block_size;
    if (idx < num_items) {
      int32_t idx0 = d_keys_in[idx] * (blockDim.x * gridDim.x) +
                     blockIdx.x * blockDim.x + threadIdx.x;
      offset[idx] = bucket_offset[idx0];
      bucket_offset[idx0]++;
    }
  }
}

#if 0
void update_indices_ptr(const int32_t *d_keys_in, const int32_t *indices_ptr_in,
                        const int32_t *offset, const int32_t *exclusive_cumsum,
                        int32_t *indices_ptr_out, int32_t num_items) {
  int32_t num_items_per_thread = num_items / THREAD_NUM;
  for (int32_t i = 0; i < THREAD_NUM; i++) {
    for (int32_t j = 0; j < num_items_per_thread; j++) {
      int32_t idx0 = j + i * num_items_per_thread;
      int32_t idx = offset[idx0] + exclusive_cumsum[d_keys_in[idx0] * THREAD_NUM + i];
      indices_ptr_out[idx] = indices_ptr_in[idx0];
      bfe_keys_out[idx] = bfe_keys[idx0];
    }
  }
}

void sort_pairs_loop(const int32_t *d_keys_in, int32_t *indices_ptr_in,
                     int32_t *indices_ptr_out, int32_t num_items) {
  put_numbers_into_bucket(d_keys_in, offset, bucket_offset, num_items);
  calc_exclusive_cumsum((int32_t *)bucket_offset, (int32_t *)exclusive_cumsum,
                        BUCKET_SIZE * THREAD_NUM);
  update_indices_ptr(d_keys_in, indices_ptr_in, offset, (int32_t *)exclusive_cumsum,
                     indices_ptr_out, num_items);
}

template <typename ValueT>
void post_process(const ValueT *d_values_in, ValueT *d_values_out,
                  int32_t *indices_ptr, int32_t *indices_ptr_out,
                  int32_t num_items) {
  for (int32_t i = 0; i < num_items; i++) {
    indices_ptr_out[i] = indices_ptr[i];
    d_values_out[i] = d_values_in[indices_ptr[i]];
  }
}

template <typename KeyT, typename ValueT, bool is_descend>
void sort_pairs(const ValueT *d_values_in, ValueT *d_values_out,
                int32_t *indices_ptr, int32_t num_items) {
  int32_t loop_count = sizeof(KeyT) * 8 / BUCKET_WIDTH;
  KeyT *keys;
  hipMalloc(&keys, sizeof(KeyT) * num_items);
  KeyT *bfe_keys;
  hipMalloc(&bfe_keys, sizeof(KeyT) * num_items);
  int32_t *offset;
  hipMalloc(&offset, sizeof(int32_t) * num_items);
  int32_t *bucket_offset;
  hipMalloc(&offset, sizeof(int32_t) * BUCKET_SIZE * THREAD_NUM);
  int32_t *exclusive_cumsum;
  hipMalloc(&exclusive_cumsum, sizeof(int32_t) * BUCKET_SIZE * THREAD_NUM);
  int32_t* indices_ptr_backup;
  hipMalloc(&indices_ptr_backup, sizeof(int32_t) * TEST_INPUT_NUM);

  prepare_keys<KeyT, ValueT, is_descend>(d_values_in, keys, num_items);
  prepare_indices(indices[0], num_items);
  for (int32_t i = 0; i < loop_count; i++) {
    int32_t begin_bit = (i)*BUCKET_WIDTH;
    extract_keys(keys, bfe_keys, indices[i % 2], num_items, begin_bit,
                 BUCKET_WIDTH);
    sort_pairs_loop(bfe_keys, indices[i % 2], indices[(i + 1) % 2], num_items);
    memset(bucket_offset, 0, sizeof(int32_t) * BUCKET_SIZE * THREAD_NUM);
  }
  post_process(d_values_in, d_values_out, indices[0], indices_ptr, num_items);

  hipFree(keys);
  hipFree(bfe_keys);
  hipFree(offset);
  hipFree(bucket_offset);
  hipFree(exclusive_cumsum);
  hipFree(indices_ptr_backup);
}
#endif

void test_sort() {
  DType *input_gpu;
  hipMalloc(&input_gpu, sizeof(float) * TEST_INPUT_NUM);
  DType *output_gpu;
  hipMalloc(&output_gpu, sizeof(float) * TEST_INPUT_NUM);
  int32_t *indices_ptr_out_gpu;
  hipMalloc(&indices_ptr_out_gpu, sizeof(int32_t) * TEST_INPUT_NUM);

  hipMemcpy(input_gpu, input, sizeof(float) * TEST_INPUT_NUM,
             hipMemcpyHostToDevice);

  // dim3 threadsPerBlock(THREAD_NUM);
  // dim3 numBlocks(2);
  // sort_pairs<int32_t, float, false>(input, output, indices_ptr_out,
  //                                   TEST_INPUT_NUM);

  hipMemcpy(output, output_gpu, sizeof(float) * TEST_INPUT_NUM,
             hipMemcpyDeviceToHost);
  hipMemcpy(indices_ptr_out, indices_ptr_out_gpu,
             sizeof(int32_t) * TEST_INPUT_NUM, hipMemcpyDeviceToHost);

  hipFree(input_gpu);
  hipFree(output_gpu);
  hipFree(indices_ptr_out);
}

int main() {
  // test_key<<<1, 2>>>();
  // test_sort();
  // test_prepare_keys();
  // test_prepare_indices();
  // test_extract_keys();
  // test_put_numbers_into_bucket();
  test_calc_exclusive_cumsum();
  hipDeviceSynchronize();
  return 0;
}
